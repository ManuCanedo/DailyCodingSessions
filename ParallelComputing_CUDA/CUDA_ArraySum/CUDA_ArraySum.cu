﻿
#include "hip/hip_runtime.h"


#include <iostream>
#include <memory>
#include <math.h>
#include <chrono>

// Timer class to benchmark the results
class Timer
{
public:
    Timer() : m_StartTimepoint(std::chrono::high_resolution_clock::now()) {}
    ~Timer() { Stop(); }

    void Stop()
    {
        const auto endTimepoint = std::chrono::high_resolution_clock::now();
        const auto start = std::chrono::time_point_cast<std::chrono::microseconds>(m_StartTimepoint).time_since_epoch().count();
        const auto end = std::chrono::time_point_cast<std::chrono::microseconds>(endTimepoint).time_since_epoch().count();
        const double ms = (end - start) * 0.001;
        std::cout << "Execution time: " << ms << "ms\n";
    }

private:
    const std::chrono::time_point<std::chrono::high_resolution_clock> m_StartTimepoint;
};

// CUDA add function
__global__
void addCUDA(int N, float* x, float* y)
{
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i < N) y[i] += x[i];
}

// Standard CPU add function
void addCPU(int N, float* x, float* y)
{
    for (int i = 0; i < N; i++)
        y[i] += x[i];
}

int main()
{
    int N = 1 << 20;

    // Standard CPU array sum
    float* x = new float[N], * y = new float[N];

    for (int i = 0; i < N; ++i)
    {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    {
        Timer CPU;
        addCPU(N, x, y);
    }

    // Free Memory
    delete[] x;
    delete[] y;
    x = y = nullptr;

    // CUDA array sum
    // Allocate Unified Memory
    hipMallocManaged(&x, N * sizeof(float));
    hipMallocManaged(&y, N * sizeof(float));

    // Initialize x and y arrays
    for (int i = 0; i < N; ++i)
    {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    {
        Timer CUDA;
        // Run kernel on 1M elements on the CPU
        addCUDA <<<N/1024, 1024>>> (N, x, y);
        // Wait for GPU to finish before assessing the result
        hipDeviceSynchronize();
    }

    for (int i = 0; i < N; ++i)
        if (y[i] != 3)
            std::cout << "Oops!";

    // Free Memory
    hipFree(x);
    hipFree(y);

    return 0;
}